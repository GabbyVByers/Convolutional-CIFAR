
#include <SFML/System.hpp>
#include <SFML/Window.hpp>
#include <SFML/Graphics.hpp>
#include <SFML/Audio.hpp>
#include <SFML/Network.hpp>

int main()
{
    sf::RenderWindow window;
    window.create(sf::VideoMode({ 800, 800 }), "My Window");

    while (window.isOpen()) {
        while (const std::optional event = window.pollEvent()) {
            if (event->is<sf::Event::Closed>())
                window.close();
        }

        window.clear(sf::Color::Blue);
        window.display();
    }

    return 0;
}
